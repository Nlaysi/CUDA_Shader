
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h> 
#include <stdlib.h> 
#include <malloc.h>

struct Vector
{
	double x;
	double y;
	double z;
};

struct Object
{
	unsigned char Type;
	unsigned char R;
	unsigned char G;
	unsigned char B;
	unsigned char atr1;
	unsigned char atr2;
	unsigned char atr3;
	unsigned char atr4;
	struct Vector position;
	struct Vector size;
	double r;
};

union Data {
	struct Object obj;
	unsigned char bytes[64];
};

struct DistanceDate {
	double distanceValue;
	unsigned char r, g, b;
};

__shared__ double lightIntensity, lightSize;
__shared__ Data* objects;
__shared__ int n;

extern "C"
__device__ Vector mul(Vector a, double value) {
	Vector v = { a.x * value, a.y * value, a.z * value };
	return v;
}

extern "C"
__device__ Vector sum(Vector a, Vector b) {
	Vector v = { a.x + b.x, a.y + b.y, a.z + b.z };
	return v;
}

extern "C"
__device__ Vector sub(Vector a, Vector b) {
	Vector v = { a.x - b.x, a.y - b.y, a.z - b.z };
	return v;
}

extern "C"
__device__ double dot(Vector a, Vector b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

extern "C"
__device__ double distance(Vector a, Vector b) {
	return sqrt((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y) + (a.z - b.z) * (a.z - b.z));
}

extern "C"
__device__ Vector normalize(Vector a) {
	Vector n = { 0, 0, 0 };
	double l = distance(a, n);
	Vector v = { a.x / l, a.y / l, a.z / l };
	return v;
}

extern "C"
__device__ double smin(double a, double b, double k)
{
	double m;
	if ((0.5 + 0.5 * (b - a) / k) > 1) 
		m = 1;
	else
		m = (0.5 + 0.5 * (b - a) / k);
	double h;
	if (m > 0)
		h = (0.5 + 0.5 * (b - a) / k);
	else
		h = 0;
	return a * h + b * (1 - h) - k * h * (1.0 - h);
}

extern "C"
__device__ double cylinder(Vector p, Object obj)
{
	Vector pa = sub(p, obj.position);
	Vector ba = sub(obj.size, obj.position);
	double baba = dot(ba, ba);
	double paba = dot(pa, ba);


	float x = distance(sub(mul(pa, baba), mul(ba, paba)), { 1,1,1 }) - obj.r * baba;

	float y = abs(paba - baba * 0.5) - baba * 0.5;
	float x2 = x * x;
	float y2 = y * y * baba;
	float d = (max(x, y) < 0.0) ? -min(x2, y2) : (((x > 0.0) ? x2 : 0.0) + ((y > 0.0) ? y2 : 0.0));

	float t = 0;
	if (d > 0)
	{
		t = 1;
	}
	if (d < 0)
	{
		t = -1;
	}

	return t * sqrt(abs(d)) / baba;
}

extern "C"
__device__ double cube(Object obj, Vector p) {

	Vector d = sub({ abs(p.x - obj.position.x), abs(p.y - obj.position.y), abs(p.z - obj.position.z) }, { obj.size.x, obj.size.y, obj.size.z});

	double insideDistance = min(max(d.x, max(d.y, d.z)), (double)0);

	if (d.x < 0.0)
		d.x = 0.0;
	if (d.y < 0.0)
		d.y = 0.0;
	if (d.z < 0.0)
		d.z = 0.0;

	double outsideDistance = sqrt(d.x * d.x + d.y * d.y + d.z * d.z);

	return insideDistance + outsideDistance;
}


__device__ double TriPrism(Vector p, Object obj)
{
	Vector q = { abs(p.x), abs(p.y), abs(p.z) };
	return max(q.z - obj.position.y, max(q.x * 0.866025 + p.y * 0.5, -p.y) - obj.position.x * 0.5);
}

extern "C"
__device__ double distanceByType(Object obj, Vector p) {
	switch (obj.Type) {
		case 1:
			return distance(p, obj.position) - obj.r;
			break;
		case 2:
			return distance(p, obj.position) - obj.r;
			break;
		case 3:
			return -(distance(p, obj.position) - obj.r);
			break;
		case 4:
			return cube(obj, p);
			break;
		case 5:
			return cube(obj, p);
			break;
		case 6:
			return -cube(obj, p);
			break;
		case 7:
			return cylinder(p, obj);
			break;
		case 8:
			return cylinder(p, obj);
			break;
		case 9:
			return -cylinder(p, obj);
			break;
		case 10:
			return TriPrism(p, obj);
			break;
		case 11:
			return TriPrism(p, obj);
			break;
		case 12:
			return -TriPrism(p, obj);
			break;
		default:
		break;
	}
}

extern "C"
__device__ DistanceDate getDist(Vector p) {
	union Data* data;
	data = objects;
	DistanceDate value;
	value.distanceValue = distanceByType(data->obj, p);
	value.r = data->obj.R;
	value.g = data->obj.G;
	value.b = data->obj.B;

	double dist = value.distanceValue;

	for (int i = 1; i < n; i++)
	{
		data = (Data*)((uintptr_t)data + sizeof(Data));
		dist = distanceByType(data->obj, p);
		if (value.distanceValue > dist && (data->obj.Type % 3 == 1 || data->obj.Type == 0)) {
			value.distanceValue = dist;
			value.r = data->obj.R;
			value.g = data->obj.G;
			value.b = data->obj.B;
		}
		if (value.distanceValue <= dist && data->obj.Type % 3 != 1) {
			value.distanceValue = dist;
			value.r = data->obj.R;
			value.g = data->obj.G;
			value.b = data->obj.B;
		}
	}
	
	return value;
}

extern "C"
__device__ Vector getNormal(Vector p)
{
	double d = getDist(p).distanceValue;
	double p1 = getDist(sub(p, { 0.001, 0, 0 })).distanceValue;
	double p2 = getDist(sub(p, { 0, 0.001, 0 })).distanceValue;
	double p3 = getDist(sub(p, { 0, 0, 0.001 })).distanceValue;
	Vector tri = { p1, p2, p3 };
	Vector di = { d, d, d };
	Vector n = sub(di, tri);
	return normalize(n);
}

extern "C"
__device__ double light(Vector p) {
	Vector lightPos = { -5, 5, -10 };
	Vector lightDir = normalize(sub(lightPos, p));
	Vector normal = getNormal(p);

	double lI = lightIntensity;
	double lS = lightSize;

	double dif = dot(normal, lightDir) * (1 - lS) * lI + lS * lI;
	return dif;
}

extern "C"
__device__ DistanceDate rayMarching(Vector ro, Vector rd) {
	Vector p = { ro.x, ro.y, ro.z };
	for (int i = 0; i < 300; i++) {
		DistanceDate d = getDist(p);
		if (d.distanceValue > 100)
			break;
		p = sum(p, mul(rd, d.distanceValue));
		if (d.distanceValue < 0.001) {
			return { light(p), d.r, d.g, d.b };
		}
	}
	return { 0, 0, 0, 0};
}

extern "C"
__global__ void draw(unsigned char* data, Data* input, int on, int widht, int height, double camX, double camY, double camZ, float vA, float hA, float lI, float lS, int pn, int pc) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = (height/pc) * pn + blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= widht || y >= height)
		return;

	objects = input;
	n = on;

	double qx = ((double)x / widht) * 2 - 1;
	double qy = ((double)y / height) * 2 - 1;

	lightIntensity = lI;
	lightSize = lS;
	
	qx = qx * ((double)widht / (double)height);
	Vector rd = { 1, qx, qy };
	rd = normalize(rd);

	double s = sin(vA);
	double c = cos(vA);
	rd = { c * rd.x + (-s) * rd.z, rd.y, s * rd.x + c * rd.z };
	s = sin(hA);
	c = cos(hA);
	rd = { c * rd.x + (-s) * rd.y, s * rd.x + c * rd.y, rd.z };

	DistanceDate col = rayMarching({camX, camY, camZ}, rd);
	if (col.distanceValue != 0) {
		data[widht * 3 * y + 3 * x    ] = (unsigned char)((double)col.r * col.distanceValue);
		data[widht * 3 * y + 3 * x + 1] = (unsigned char)((double)col.g * col.distanceValue);
		data[widht * 3 * y + 3 * x + 2] = (unsigned char)((double)col.b * col.distanceValue);
	}
	else
	{
		data[widht * 3 * y + 3 * x] = (unsigned char)((double)117 * lI);
		data[widht * 3 * y + 3 * x + 1] = (unsigned char)((double)187 * lI);
		data[widht * 3 * y + 3 * x + 2] = (unsigned char)((double)253 * lI);
	}
    
}